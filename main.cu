#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cblas.h>
#include <algorithm>
#include <iostream>
#include <vector>
#include <chrono>
#include <bits/stdc++.h>

// Define the payoff matrix (assuming Hawk-Dove game)
#define FULL_PAYOFF 20
#define HAWK_VS_DOVE_PAYOFF 15
#define DOVE_VS_HAWK_PAYOFF 5
#define DOVE_VS_DOVE_PAYOFF 10
// Defaults
int bushes = 60;
int hawks = 10;
int doves = 10;
int generations = 100;

__device__ void contestGPU(int strat1, int strat2, int &output)
{
    if (strat1 == 1 && strat2 == 1)
    { // Both players choose Hawk
      // The cost is paid prior to calling this method.
    }
    else if (strat1 == 1 && strat2 == 0)
    { // Hawk vs Dove
        output = HAWK_VS_DOVE_PAYOFF;
    }
    else if (strat1 == 0 && strat2 == 1)
    { // Dove vs Hawk
        output = DOVE_VS_HAWK_PAYOFF;
    }
    else if (strat1 == 0 && strat2 == 0)
    { // Both players choose Dove
        output = DOVE_VS_DOVE_PAYOFF;
    }
    else
    {
        output = FULL_PAYOFF;
    }
}

__global__ void hawkDoveKernel(int *strategies, int *score, int numBirds)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < numBirds)
    {
        int my_strategy = strategies[idx];
        int opponent_strategy = strategies[(idx + 1) % numBirds]; // Simple opponent selection

        // Determine score based on strategies
        contestGPU(my_strategy, opponent_strategy, score[idx]);
    }
}

enum struct Strategy : uint16_t
{
    Share = 0,
    Steal,
    Bush
};

enum struct LifeCycle : uint16_t
{
    Live = 0,
    Reproduce,
    Die
};

class Creature
{
public:
    Strategy strategy;
    int score = 0;
    LifeCycle lifecycle = LifeCycle::Live;
};

class Bird : Creature
{
public:
    Bird(Strategy input)
    {
        strategy = input;
        score = 10;
    }
    Strategy getStrategy()
    {
        return strategy;
    }
    void setStrategy(const Strategy &input)
    {
        strategy = input;
    }
    int getScore()
    {
        return score;
    }
    void setScore(const int &input)
    {
        score = input;
    }
    LifeCycle getLifeCycle()
    {
        return lifecycle;
    }
    void setLifeCycle(const LifeCycle &input)
    {
        lifecycle = input;
    }
};

int strategyToInt(Strategy input)
{
    if (input == Strategy::Steal)
    {
        return 0;
    }
    else if (input == Strategy::Share)
    {
        return 1;
    }
    else
    {
        return 2;
    }
}

void whatAmI(Bird &testBird)
{
    if (testBird.getStrategy() == Strategy::Steal)
    {
        std::cout << "I am a Hawk." << std::endl;
    }
    else if (testBird.getStrategy() == Strategy::Share)
    {
        std::cout << "I am a Dove." << std::endl;
    }
    else
    {
        std::cout << "I am Unknown." << std::endl;
    }
}

void whatIsMyLifeCycle(Bird &testBird)
{
    if (testBird.getLifeCycle() == LifeCycle::Live)
    {
        std::cout << "I will Live." << std::endl;
    }
    else if (testBird.getLifeCycle() == LifeCycle::Die)
    {
        std::cout << "I will Die." << std::endl;
    }
    else if (testBird.getLifeCycle() == LifeCycle::Reproduce)
    {
        std::cout << "I will Reproduce." << std::endl;
    }
}

void countBirds(std::vector<Bird> &birdArray, int arraySize, int &hawks,
                int &doves)
{
    doves = 0;
    hawks = 0;

    for (int i = 0; i < arraySize; i++)
    {
        if (birdArray[i].getStrategy() == Strategy::Share)
            doves++;
        else
            hawks++;
    }
}

void contest(Bird &first, Bird &second, int v, int c)
{
    // Test logs left in.
    if (first.getStrategy() == Strategy::Share && second.getStrategy() == Strategy::Share)
    {
        // std::cout << "Both are Doves, share resources \n";
        first.setScore(first.getScore() + (v * .5));
        second.setScore(second.getScore() + (v * .5));
    }
    else if (first.getStrategy() == Strategy::Steal && second.getStrategy() == Strategy::Steal)
    {
        // std::cout << "Both are Hawks, waste energy fighting \n";
        first.setScore(first.getScore() + 0);
        second.setScore(second.getScore() + 0);
    }
    else if (first.getStrategy() == Strategy::Share && second.getStrategy() == Strategy::Steal)
    {
        // std::cout << "Dove Meets Hawk \n";
        first.setScore(first.getScore() + (v * .25));
        second.setScore(second.getScore() + (v * .75));
    }
    else if (first.getStrategy() == Strategy::Steal && second.getStrategy() == Strategy::Share)
    {
        // std::cout << "Hawk Meets Dove \n";
        first.setScore(first.getScore() + (v * .75));
        second.setScore(second.getScore() + (v * .25));
    }
    else
    {
        // std::cout << "There's a bush \n";
        first.setScore(first.getScore() + v);
        second.setScore(second.getScore() + v);
    }
}

void vectorAddMultiples(std::vector<Bird> &array, int size, Strategy toAdd)
{
    for (int i = 0; i < size; i++)
    {
        Bird newBird(toAdd);
        array.push_back(newBird);
    }
}

void runTestGpuKernel(std::vector<Bird> &birdVector, int bushes)
{
    // The cost of life
    for (unsigned i = 0; i < birdVector.size(); ++i)
        birdVector[i].setScore(birdVector[i].getScore() - 10);

    int diff = (bushes * 2) - birdVector.size();
    if (diff > 0)
        vectorAddMultiples(birdVector, diff, Strategy::Bush);

    // Shuffle to randomize the contests
    std::shuffle(std::begin(birdVector), std::end(birdVector), std::default_random_engine(std::chrono::system_clock::now().time_since_epoch().count()));

    // Initialize game parameters
    int numOfBirds = birdVector.size(); // Number of birds
    int strategies[numOfBirds];         // Array to hold strategies (0 for Dove, 1 for Hawk, 2 for Bush)
    int payoffs[numOfBirds];            // Array to hold payoffs

    // Initialize strategies randomly (for demonstration)
    for (int i = 0; i < numOfBirds; ++i)
    {
        strategies[i] = strategyToInt(birdVector[i].getStrategy());
    }

    // Allocate device memory
    int *d_strategies;
    int *d_payoffs;
    checkCudaErrors(hipMalloc(&d_strategies, numOfBirds * sizeof(int)));
    checkCudaErrors(hipMalloc(&d_payoffs, numOfBirds * sizeof(int)));

    // Copy strategies to device
    checkCudaErrors(hipMemcpy(d_strategies, strategies, numOfBirds * sizeof(int), hipMemcpyHostToDevice));

    // Define grid and block dimensions
    int blockSize = 256;
    int numBlocks = (numOfBirds + blockSize - 1) / blockSize;

    double cumulativeTime = 0.0;

    {
        hipEvent_t start_, stop_;
        checkCudaErrors(hipEventCreate(&start_));
        checkCudaErrors(hipEventCreate(&stop_));
        checkCudaErrors(hipEventRecord(start_));

        // Launch kernel
        hawkDoveKernel<<<numBlocks, blockSize>>>(d_strategies, d_payoffs, numOfBirds);

        checkCudaErrors(hipEventRecord(stop_));
        checkCudaErrors(hipEventSynchronize(stop_));

        // Copy payoffs from device
        checkCudaErrors(hipMemcpy(payoffs, d_payoffs, numOfBirds * sizeof(int), hipMemcpyDeviceToHost));

        float milliseconds = 0.0f;

        checkCudaErrors(hipEventElapsedTime(&milliseconds, start_, stop_));
        cumulativeTime += milliseconds;
        checkCudaErrors(hipEventDestroy(start_));
        checkCudaErrors(hipEventDestroy(stop_));
    }

    // Display payoffs (for demonstration)
    // printf("Payoffs:\n");
    // for (int i = 0; i < numOfBirds; ++i)
    // {
    //     printf("Player %d: %d\n", i, payoffs[i]);
    // }

    for (int i = 0; i < numOfBirds; ++i)
    {
        birdVector[i].setScore(payoffs[i]);
    }

    for (int i = birdVector.size() - 1; i >= 0; i--)
    {
        if (birdVector[i].getStrategy() == Strategy::Bush)
        {
            birdVector.erase(birdVector.begin() + i);
        }
    }

    // Free device memory
    checkCudaErrors(hipFree(d_strategies));
    checkCudaErrors(hipFree(d_payoffs));
}

void DayCycle(std::vector<Bird> &birdVector, int bushes)
{
    // The cost of life
    for (unsigned i = 0; i < birdVector.size(); ++i)
        birdVector[i].setScore(birdVector[i].getScore() - 10);

    int diff = (bushes * 2) - birdVector.size();
    if (diff > 0)
        vectorAddMultiples(birdVector, diff, Strategy::Bush);

    // Shuffle to randomize the contests
    std::shuffle(std::begin(birdVector), std::end(birdVector), std::default_random_engine(std::chrono::system_clock::now().time_since_epoch().count()));

    for (int j = 0; j < (bushes * 2); j += 2)
    {
        contest(birdVector[j], birdVector[j + 1], 20, 0);
    }

    for (int i = birdVector.size() - 1; i >= 0; i--)
    {
        if (birdVector[i].getStrategy() == Strategy::Bush)
        {
            birdVector.erase(birdVector.begin() + i);
        }
    }
}

void NightCycle(std::vector<Bird> &array)
{
    int babyDoves = 0;
    int babyHawks = 0;
    int testValue = 0;

    for (int i = 0; i < array.size(); i++)
    {
        array[i].setLifeCycle(LifeCycle::Live);

        int chance = rand() % 10 + 1;

        testValue = array[i].getScore();
        if (testValue < 10)
        {
            if (testValue < chance)
                array[i].setLifeCycle(LifeCycle::Die);
        }
        else if (testValue >= 20)
        {
            array[i].setScore(testValue - 10);
            if (array[i].getStrategy() == Strategy::Steal)
            {
                babyHawks++;
            }
            else
            {
                babyDoves++;
            }
        }
        else
        {
            if (testValue > (chance + 10))
            {
                array[i].setScore(testValue - 10);
                if (array[i].getStrategy() == Strategy::Steal)
                {
                    babyHawks++;
                }
                else
                {
                    babyDoves++;
                }
            }
        }
    }

    // Test log
    // std::cout << " Adding babies, " << babyDoves << " doves and " << babyHawks << " hawks \n";
    vectorAddMultiples(array, babyDoves, Strategy::Share);
    vectorAddMultiples(array, babyHawks, Strategy::Steal);

    for (int i = array.size() - 1; i >= 0; i--)
    {
        if (array[i].getLifeCycle() == LifeCycle::Die)
        {
            array.erase(array.begin() + i);
        }
    }
}

int runCpu()
{
    std::vector<Bird> arr;

    vectorAddMultiples(arr, doves, Strategy::Share);
    vectorAddMultiples(arr, hawks, Strategy::Steal);

    int hawksval = 0;
    int dovesval = 0;

    std::ofstream myfile;
    myfile.open("cpuoutput.csv");
    myfile << "dove,hawk\n";

    auto start = std::chrono::high_resolution_clock::now();

    // Loop function
    for (int i = 1; i <= generations; i++)
    {
        // Count birds and output birds and generation
        countBirds(arr, arr.size(), hawksval, dovesval);
        std::cout << "Generation : " << i << ". # of bushes : " << bushes << ". # of Hawks : " << hawksval << " # of Doves : " << dovesval << std::endl;

        myfile << dovesval << "," << hawksval << "\n";

        // Allow life to happen
        DayCycle(arr, bushes);
        NightCycle(arr);
    }

    auto stop = std::chrono::high_resolution_clock::now();

    myfile.close();

    auto duration =
        std::chrono::duration_cast<std::chrono::microseconds>(stop - start);

    std::cout << "Average Time taken by function: "
              << duration.count() / generations << " microseconds" << std::endl;
    return 0;
}

void printTestInfo(std::vector<Bird> &arr)
{
    std::cout << "Number of Birds " << arr.size() << std::endl;

    std::cout << "Birds: ";

    for (unsigned i = 0; i < arr.size(); ++i)
        std::cout << ' ' << arr[i].getScore();
    std::cout << '\n';
}

void runGPU()
{
    std::vector<Bird> arr;

    vectorAddMultiples(arr, doves, Strategy::Share);
    vectorAddMultiples(arr, hawks, Strategy::Steal);

    int hawksval = 0;
    int dovesval = 0;

    std::ofstream myfile;
    myfile.open("gpuoutput.csv");
    myfile << "dove,hawk\n";

    auto start = std::chrono::high_resolution_clock::now();

    // Loop function
    for (int i = 1; i <= generations; i++)
    {
        // Count birds and output birds and generation
        countBirds(arr, arr.size(), hawksval, dovesval);
        std::cout << "Generation : " << i << ". # of bushes : " << bushes << ". # of Hawks : " << hawksval << " # of Doves : " << dovesval << std::endl;

        myfile << dovesval << "," << hawksval << "\n";

        // Allow life to happen
        runTestGpuKernel(arr, bushes);
        NightCycle(arr);
    }

    auto stop = std::chrono::high_resolution_clock::now();

    myfile.close();

    auto duration =
        std::chrono::duration_cast<std::chrono::microseconds>(stop - start);

    std::cout << "Average Time taken by function: "
              << duration.count() / generations << " microseconds" << std::endl;
}

void GetIntFromRange(int lower, int upper, int &input)
{
    std::cout << "Please give me a number between " << lower << " and " << upper << ": " << std::flush;

    // First error catch. If it's not an integer, don't even let it get to bounds control
    while (!(std::cin >> input))
    {
        std::cout << "Wrong Input Type. Please try again.\n";
        std::cin.clear();
        std::cin.ignore(std::numeric_limits<std::streamsize>::max(), '\n');
    }

    // Bounds control
    while (input < lower || input > upper)
    {
        std::cout << "Out of Range. Re-enter option: ";
        std::cin.ignore(std::numeric_limits<std::streamsize>::max(), '\n');

        // Second error catch. If out of range integer was entered, and then a non-integer this second one shall catch it
        while (!(std::cin >> input))
        {
            std::cout << "Wrong Input Type. Please try again.\n";
            std::cin.clear();
            std::cin.ignore(std::numeric_limits<std::streamsize>::max(), '\n');
        }
    }
}

int main()
{
    // std::cout << "Please provide bushes to seed the environment; " << std::endl;
    // GetIntFromRange(1, 100, bushes);
    // std::cout << std::endl
    //           << "Please provide hawks to seed the environment; " << std::endl;
    // GetIntFromRange(1, 100, hawks);
    // std::cout << std::endl
    //           << "Please provide doves to seed the environment; " << std::endl;
    // GetIntFromRange(1, 100, doves);
    // std::cout << std::endl
    //           << "Please provide generations to run; " << std::endl;
    // GetIntFromRange(1, 1000, generations);

    // printf("Bushes, hawks, doves, and generations are now %i, %i, %i, %i \n", bushes, hawks, doves, generations);

    std::cout << "Runing CPU" << std::endl;
    runCpu();

    std::cout << "\n Runing GPU" << std::endl;
    runGPU();

    return 0;
}
